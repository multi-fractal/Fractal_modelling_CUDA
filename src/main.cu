#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void addArrays(int* a, int* b, int* result, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        result[i] = a[i] + b[i];
    }
}

int main() {
    const int N = 10;
    int h_a[N], h_b[N], h_result[N];

    // Заполняем массивы
    for (int i = 0; i < N; ++i) {
        h_a[i] = i;
        h_b[i] = 2 * i;
    }

    // Выделяем память на GPU
    int *d_a, *d_b, *d_result;
    hipMalloc((void**)&d_a, N * sizeof(int));
    hipMalloc((void**)&d_b, N * sizeof(int));
    hipMalloc((void**)&d_result, N * sizeof(int));

    // Копируем данные на GPU
    hipMemcpy(d_a, h_a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(int), hipMemcpyHostToDevice);

    // Запускаем ядро
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    addArrays<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_result, N);

    // Копируем результат обратно в host-память
    hipMemcpy(h_result, d_result, N * sizeof(int), hipMemcpyDeviceToHost);

    // Выводим результат
    printf("Result of a + b:\n");
    for (int i = 0; i < N; ++i) {
        printf("%d + %d = %d\n", h_a[i], h_b[i], h_result[i]);
    }

    // Освобождаем память
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);

    return 0;
}
